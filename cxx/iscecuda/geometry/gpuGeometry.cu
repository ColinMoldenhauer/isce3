#include "hip/hip_runtime.h"
#include "gpuGeometry.h"

#include <isce/core/Basis.h>
#include <isce/core/Ellipsoid.h>
#include <isce/core/LookSide.h>
#include <isce/core/Orbit.h>
#include <isce/core/Pixel.h>
#include <isce/cuda/core/Orbit.h>
#include <isce/cuda/core/OrbitView.h>
#include <isce/cuda/core/gpuLUT1d.h>
#include <isce/cuda/core/gpuLUT2d.h>
#include <isce/cuda/except/Error.h>
#include <isce/cuda/geometry/gpuDEMInterpolator.h>
#include <isce/geometry/detail/Geo2Rdr.h>
#include <isce/geometry/detail/Rdr2Geo.h>

namespace detail = isce::geometry::detail;

using isce::core::Basis;
using isce::core::LookSide;
using isce::core::OrbitInterpBorderMode;
using isce::core::Vec3;
using isce::error::ErrorCode;

namespace isce { namespace cuda { namespace geometry {

CUDA_DEV
int rdr2geo(const isce::core::Pixel& pixel, const Basis& TCNbasis,
            const Vec3& pos, const Vec3& vel,
            const isce::core::Ellipsoid& ellipsoid,
            const gpuDEMInterpolator& demInterp, Vec3& targetLLH, LookSide side,
            double threshold, int maxIter, int extraIter)
{
    double h0 = targetLLH[2];
    detail::Rdr2GeoParams params = {threshold, maxIter, extraIter};
    auto status = detail::rdr2geo(&targetLLH, pixel, TCNbasis, pos, vel,
                                  demInterp, ellipsoid, side, h0, params);
    return (status == ErrorCode::Success);
}

__device__ int rdr2geo(double aztime, double slant_range, double doppler,
                       const isce::cuda::core::OrbitView& orbit,
                       const isce::core::Ellipsoid& ellipsoid,
                       const gpuDEMInterpolator& dem_interp, Vec3& target_llh,
                       double wvl, LookSide side, double threshold,
                       int max_iter, int extra_iter)
{
    double h0 = target_llh[2];
    detail::Rdr2GeoParams params = {threshold, max_iter, extra_iter};
    auto status =
            detail::rdr2geo(&target_llh, aztime, slant_range, doppler, orbit,
                            dem_interp, ellipsoid, wvl, side, h0, params);
    return (status == ErrorCode::Success);
}

CUDA_DEV
int geo2rdr(const Vec3& inputLLH, const isce::core::Ellipsoid& ellipsoid,
            const isce::cuda::core::OrbitView& orbit,
            const isce::cuda::core::gpuLUT1d<double>& doppler,
            double* aztime_result, double* slantRange_result, double wavelength,
            LookSide side, double threshold, int maxIter, double deltaRange)
{

    // Cartesian type local variables
    // Temp local variables for results
    double aztime, slantRange;

    // Convert LLH to XYZ
    const Vec3 inputXYZ = ellipsoid.lonLatToXyz(inputLLH);

    // Pre-compute scale factor for doppler
    const double dopscale = 0.5 * wavelength;

    // Use mid-orbit epoch as initial guess
    aztime = orbit.midTime();

    // Begin iterations
    int converged = 0;
    double slantRange_old = 0.0;
    for (int i = 0; i < maxIter; ++i) {

        // Interpolate the orbit to current estimate of azimuth time
        Vec3 pos, vel;
        orbit.interpolate(&pos, &vel, aztime, OrbitInterpBorderMode::FillNaN);

        // Compute slant range from satellite to ground point
        const Vec3 dr = inputXYZ - pos;
        slantRange = dr.norm();

        // Check look side
        // (Left && positive) || (Right && negative)
        if ((side == LookSide::Right) ^ (dr.cross(vel).dot(pos) > 0)) {
            *slantRange_result = slantRange;
            *aztime_result = aztime;
            return converged;
        }

        // Check convergence
        if (std::abs(slantRange - slantRange_old) < threshold) {
            converged = 1;
            *slantRange_result = slantRange;
            *aztime_result = aztime;
            return converged;
        } else {
            slantRange_old = slantRange;
        }

        // Compute doppler
        const double dopfact = dr.dot(vel);
        const double fdop = doppler.eval(slantRange) * dopscale;
        // Use forward difference to compute doppler derivative
        const double fdopder =
                (doppler.eval(slantRange + deltaRange) * dopscale - fdop) /
                deltaRange;

        // Evaluate cost function and its derivative
        const double fn = dopfact - fdop * slantRange;
        const double c1 = -vel.dot(vel);
        const double c2 = (fdop / slantRange) + fdopder;
        const double fnprime = c1 + c2 * dopfact;

        // Update guess for azimuth time
        aztime -= fn / fnprime;
    }

    // If we reach this point, no convergence for specified threshold
    *slantRange_result = slantRange;
    *aztime_result = aztime;
    return converged;
}

CUDA_DEV int geo2rdr(const isce::core::Vec3& inputLLH,
                     const isce::core::Ellipsoid& ellipsoid,
                     const isce::cuda::core::OrbitView& orbit,
                     const isce::cuda::core::gpuLUT2d<double>& doppler,
                     double* aztime, double* slantRange, double wavelength,
                     isce::core::LookSide side, double threshold, int maxIter,
                     double deltaRange)
{
    double t0 = *aztime;
    detail::Geo2RdrParams params = {threshold, maxIter, deltaRange};
    auto status =
            detail::geo2rdr(aztime, slantRange, inputLLH, ellipsoid, orbit,
                            doppler, wavelength, side, t0, params);
    return (status == ErrorCode::Success);
}

}}} // namespace isce::cuda::geometry

// Create ProjectionBase pointer on the device (meant to be run by a single
// thread)
__global__ void createProjection(isce::cuda::core::ProjectionBase** proj,
                                 int epsgCode)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        (*proj) = isce::cuda::core::createProj(epsgCode);
    }
}

// Delete ProjectionBase pointer on the device (meant to be run by a single
// thread)
__global__ void deleteProjection(isce::cuda::core::ProjectionBase** proj)
{
    delete *proj;
}

namespace isce { namespace cuda { namespace geometry {

// Helper kernel to call device-side rdr2geo
__global__ void rdr2geo_d(const isce::core::Pixel pixel, const Basis TCNbasis,
                          const Vec3 pos, const Vec3 vel,
                          const isce::core::Ellipsoid ellipsoid,
                          gpuDEMInterpolator demInterp, Vec3* targetLLH,
                          LookSide side, double threshold, int maxIter,
                          int extraIter, int* resultcode)
{

    // Call device function
    *resultcode = rdr2geo(pixel, TCNbasis, pos, vel, ellipsoid, demInterp,
                          *targetLLH, side, threshold, maxIter, extraIter);
}

// Host radar->geo to test underlying functions in a single-threaded context
CUDA_HOST
int rdr2geo_h(const isce::core::Pixel& pixel, const Basis& basis,
              const Vec3& pos, const Vec3& vel,
              const isce::core::Ellipsoid& ellipsoid,
              isce::geometry::DEMInterpolator& demInterp, Vec3& llh,
              LookSide side, double threshold, int maxIter, int extraIter)
{

    // Make GPU objects
    gpuDEMInterpolator gpu_demInterp(demInterp);

    // Allocate device memory
    Vec3* llh_d;
    int* resultcode_d;
    hipMalloc((double**) &llh_d, 3 * sizeof(double));
    hipMalloc((int**) &resultcode_d, sizeof(int));

    // Copy initial values
    hipMemcpy(llh_d, llh.data(), 3 * sizeof(double), hipMemcpyHostToDevice);

    // DEM interpolator initializes its projection and interpolator
    gpu_demInterp.initProjInterp();

    // Run the rdr2geo on the GPU
    dim3 grid(1), block(1);
    rdr2geo_d<<<grid, block>>>(pixel, basis, pos, vel, ellipsoid, gpu_demInterp,
                               llh_d, side, threshold, maxIter, extraIter,
                               resultcode_d);

    // Check for any kernel errors
    checkCudaErrors(hipPeekAtLastError());

    // Delete projection pointer on device
    gpu_demInterp.finalizeProjInterp();

    // Copy the resulting llh back to the CPU
    int resultcode;
    checkCudaErrors(hipMemcpy(llh.data(), llh_d, 3 * sizeof(double),
                               hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(&resultcode, resultcode_d, sizeof(int),
                               hipMemcpyDeviceToHost));

    // Free memory
    checkCudaErrors(hipFree(llh_d));
    checkCudaErrors(hipFree(resultcode_d));

    // Return result code
    return resultcode;
}

// Helper kernel to call device-side geo2rdr
__global__ void geo2rdr_d(const Vec3 llh, isce::core::Ellipsoid ellps,
                          isce::cuda::core::OrbitView orbit,
                          isce::cuda::core::gpuLUT1d<double> doppler,
                          double* aztime, double* slantRange, double wavelength,
                          LookSide side, double threshold, int maxIter,
                          double deltaRange, int* resultcode)
{

    // Call device function
    *resultcode = geo2rdr(llh, ellps, orbit, doppler, aztime, slantRange,
                          wavelength, side, threshold, maxIter, deltaRange);
}

// Host geo->radar to test underlying functions in a single-threaded context
CUDA_HOST
int geo2rdr_h(const cartesian_t& llh, const isce::core::Ellipsoid& ellps,
              const isce::core::Orbit& orbit,
              const isce::core::LUT1d<double>& doppler, double& aztime,
              double& slantRange, double wavelength, LookSide side,
              double threshold, int maxIter, double deltaRange)
{

    // Make GPU objects
    isce::core::Ellipsoid gpu_ellps(ellps);
    isce::cuda::core::Orbit gpu_orbit(orbit);
    isce::cuda::core::gpuLUT1d<double> gpu_doppler(doppler);

    // Allocate necessary device memory
    double *llh_d, *aztime_d, *slantRange_d;
    int* resultcode_d;
    hipMalloc((double**) &llh_d, 3 * sizeof(double));
    hipMalloc((double**) &aztime_d, sizeof(double));
    hipMalloc((double**) &slantRange_d, sizeof(double));
    hipMalloc((int**) &resultcode_d, sizeof(int));

    // Copy input values
    hipMemcpy(llh_d, llh.data(), 3 * sizeof(double), hipMemcpyHostToDevice);

    // Run geo2rdr on the GPU
    dim3 grid(1), block(1);
    geo2rdr_d<<<grid, block>>>(llh, gpu_ellps, gpu_orbit, gpu_doppler, aztime_d,
                               slantRange_d, wavelength, side, threshold,
                               maxIter, deltaRange, resultcode_d);

    // Copy results to CPU and return any error code
    int resultcode;
    hipMemcpy(&aztime, aztime_d, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&slantRange, slantRange_d, sizeof(double),
               hipMemcpyDeviceToHost);
    hipMemcpy(&resultcode, resultcode_d, sizeof(int), hipMemcpyDeviceToHost);

    // Free memory
    hipFree(llh_d);
    hipFree(aztime_d);
    hipFree(slantRange_d);
    hipFree(resultcode_d);

    // Return error code
    return resultcode;
}

}}} // namespace isce::cuda::geometry

#include "CufftWrapper.h"

#include <isce/cuda/except/Error.h>

namespace isce { namespace cuda { namespace fft { namespace detail {

template<>
void executePlan<HIPFFT_FORWARD, float>(hipfftHandle plan, void * in, void * out, hipfftType type)
{
    if (type == HIPFFT_C2C) {
        checkCudaErrors( hipfftExecC2C(
                plan,
                reinterpret_cast<hipfftComplex *>(in),
                reinterpret_cast<hipfftComplex *>(out),
                HIPFFT_FORWARD) );
    }
    else { // HIPFFT_R2C
        checkCudaErrors( hipfftExecR2C(
                plan,
                reinterpret_cast<hipfftReal *>(in),
                reinterpret_cast<hipfftComplex *>(out)) );
    }
}

template<>
void executePlan<HIPFFT_FORWARD, double>(hipfftHandle plan, void * in, void * out, hipfftType type)
{
    if (type == HIPFFT_Z2Z) {
        checkCudaErrors( hipfftExecZ2Z(
                plan,
                reinterpret_cast<hipfftDoubleComplex *>(in),
                reinterpret_cast<hipfftDoubleComplex *>(out),
                HIPFFT_FORWARD) );
    }
    else { // HIPFFT_D2Z
        checkCudaErrors( hipfftExecD2Z(
                plan,
                reinterpret_cast<hipfftDoubleReal *>(in),
                reinterpret_cast<hipfftDoubleComplex *>(out)) );
    }
}

template<>
void executePlan<HIPFFT_BACKWARD, float>(hipfftHandle plan, void * in, void * out, hipfftType type)
{
    if (type == HIPFFT_C2C) {
        checkCudaErrors( hipfftExecC2C(
                plan,
                reinterpret_cast<hipfftComplex *>(in),
                reinterpret_cast<hipfftComplex *>(out),
                HIPFFT_BACKWARD) );
    }
    else { // HIPFFT_C2R
        checkCudaErrors( hipfftExecC2R(
                plan,
                reinterpret_cast<hipfftComplex *>(in),
                reinterpret_cast<hipfftReal *>(out)) );
    }
}

template<>
void executePlan<HIPFFT_BACKWARD, double>(hipfftHandle plan, void * in, void * out, hipfftType type)
{
    if (type == HIPFFT_Z2Z) {
        checkCudaErrors( hipfftExecZ2Z(
                plan,
                reinterpret_cast<hipfftDoubleComplex *>(in),
                reinterpret_cast<hipfftDoubleComplex *>(out),
                HIPFFT_BACKWARD) );
    }
    else { // HIPFFT_Z2D
        checkCudaErrors( hipfftExecZ2D(
                plan,
                reinterpret_cast<hipfftDoubleComplex *>(in),
                reinterpret_cast<hipfftDoubleReal *>(out)) );
    }
}

}}}}

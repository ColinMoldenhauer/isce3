#include "Device.h"

#include <isce3/cuda/except/Error.h>
#include <isce3/except/Error.h>

namespace isce3 { namespace cuda { namespace core {

static hipDeviceProp_t getDeviceProperties(int id)
{
    hipDeviceProp_t props;
    checkCudaErrors(hipGetDeviceProperties(&props, id));
    return props;
}

Device::Device(int id) : _id(id)
{
    const int count = getDeviceCount();
    if (id < 0 or id >= count) {
        const std::string errmsg =
                "invalid CUDA device index - " + std::to_string(id);
        throw isce3::except::InvalidArgument(ISCE_SRCINFO(), errmsg);
    }
}

std::string Device::name() const { return getDeviceProperties(id()).name; }

size_t Device::totalGlobalMem() const
{
    return getDeviceProperties(id()).totalGlobalMem;
}

ComputeCapability Device::computeCapability() const
{
    const auto props = getDeviceProperties(id());
    return {props.major, props.minor};
}

int getDeviceCount()
{
    int count = -1;
    checkCudaErrors(hipGetDeviceCount(&count));
    return count;
}

Device getDevice()
{
    int device = -1;
    checkCudaErrors(hipGetDevice(&device));
    return device;
}

void setDevice(Device d) { checkCudaErrors(hipSetDevice(d.id())); }

}}} // namespace isce3::cuda::core

#include "Orbit.h"

#include <isce3/core/Common.h>
#include <isce3/core/detail/BuildOrbit.h>
#include <isce3/error/ErrorCode.h>
#include <isce3/except/Error.h>

#include <isce3/cuda/except/Error.h>

#include "OrbitView.h"

using isce3::core::DateTime;
using isce3::core::OrbitInterpMethod;
using isce3::core::OrbitInterpBorderMode;
using isce3::core::StateVector;
using isce3::core::TimeDelta;
using isce3::core::Vec3;
using isce3::error::ErrorCode;
using isce3::error::getErrorString;

using HostOrbit = isce3::core::Orbit;

namespace isce3 { namespace cuda { namespace core {

Orbit::Orbit(const HostOrbit & orbit)
:
    _reference_epoch(orbit.referenceEpoch()),
    _time(orbit.time()),
    _position(orbit.position()),
    _velocity(orbit.velocity()),
    _interp_method(orbit.interpMethod())
{}

Orbit::Orbit(const std::vector<StateVector> & statevecs,
             OrbitInterpMethod interp_method)
:
    Orbit(HostOrbit(statevecs, interp_method))
{}

Orbit::Orbit(const std::vector<StateVector> & statevecs,
             const DateTime & reference_epoch,
             OrbitInterpMethod interp_method)
:
    Orbit(HostOrbit(statevecs, reference_epoch, interp_method))
{}

// convenience function to get device vector data pointer
template<typename T>
constexpr
const T * dptr(const thrust::device_vector<T> & v) { return v.data().get(); }

// convenience function to get device vector data pointer
template<typename T>
constexpr
T * dptr(thrust::device_vector<T> & v) { return v.data().get(); }

// copy device vector to std::vector
template<typename T>
inline
std::vector<T> copyToHost(const thrust::device_vector<T> & d)
{
    std::vector<T> h(d.size());

    if (d.size() != 0) {
        T * dst = h.data();
        const T * src = dptr(d);
        std::size_t count = d.size() * sizeof(T);
        checkCudaErrors( hipMemcpy(dst, src, count, hipMemcpyDeviceToHost) );
    }

    return h;
}

std::vector<StateVector> Orbit::getStateVectors() const
{
    // copy to host
    std::vector<Vec3> pos = copyToHost(_position);
    std::vector<Vec3> vel = copyToHost(_velocity);

    // convert to state vectors
    std::vector<StateVector> statevecs(size());
    for (int i = 0; i < size(); ++i) {
        statevecs[i].datetime = _reference_epoch + TimeDelta(_time[i]);
        statevecs[i].position = _position[i];
        statevecs[i].velocity = _velocity[i];
    }

    return statevecs;
}

void Orbit::setStateVectors(const std::vector<StateVector> & statevecs)
{
    _time = isce3::core::detail::getOrbitTime(statevecs, _reference_epoch);
    _position = isce3::core::detail::getOrbitPosition(statevecs);
    _velocity = isce3::core::detail::getOrbitVelocity(statevecs);
}

void Orbit::referenceEpoch(const DateTime & reference_epoch)
{
    DateTime old_refepoch = _reference_epoch;
    double old_starttime = _time.first();

    double dt = (old_refepoch - reference_epoch).getTotalSeconds();
    double starttime = old_starttime + dt;

    _time.first(starttime);
    _reference_epoch = reference_epoch;
}

static
CUDA_GLOBAL
void interpOrbit(Vec3 * position,
                 Vec3 * velocity,
                 OrbitView orbit,
                 double t,
                 OrbitInterpBorderMode border_mode,
                 ErrorCode * status)
{
    // no bounds checking - assume single-threaded execution
    ErrorCode ret = orbit.interpolate(position, velocity, t, border_mode);
    if (status && ret != ErrorCode::Success) {
        *status = ret;
    }
}

ErrorCode Orbit::interpolate(Vec3 * position,
                        Vec3 * velocity,
                        double t,
                        OrbitInterpBorderMode border_mode) const
{
    // init device memory for results & status code
    thrust::device_vector<Vec3> d_pos(1), d_vel(1);
    thrust::device_vector<ErrorCode> d_stat(1, ErrorCode::Success);

    // launch kernel, check for launch & execution errors
    interpOrbit<<<1, 1>>>(dptr(d_pos), dptr(d_vel), *this, t, border_mode, dptr(d_stat));
    checkCudaErrors( hipPeekAtLastError() );
    checkCudaErrors( hipStreamSynchronize(0) );

    // check return code
    ErrorCode status = d_stat[0];
    if (status != ErrorCode::Success and
            border_mode == OrbitInterpBorderMode::Error) {

        std::string errmsg = getErrorString(status);
        throw isce3::except::OutOfRange(ISCE_SRCINFO(), errmsg);
    }

    if (position) { *position = d_pos[0]; }
    if (velocity) { *velocity = d_vel[0]; }

    return status;
}

bool operator==(const Orbit & lhs, const Orbit & rhs)
{
    return lhs.referenceEpoch() == rhs.referenceEpoch() &&
           lhs.time() == rhs.time() &&
           lhs.position() == rhs.position() &&
           lhs.velocity() == rhs.velocity() &&
           lhs.interpMethod() == rhs.interpMethod();
}

bool operator!=(const Orbit & lhs, const Orbit & rhs)
{
    return !(lhs == rhs);
}

}}}


#include <hip/hip_runtime.h>
#include <cstdio>

int main()
{
    // get number of CUDA devices
    hipError_t status;
    int count = 0;
    status = hipGetDeviceCount(&count);
    if (status != hipSuccess) {
        return 1;
    }

    // print compute capability for each device
    hipDeviceProp_t prop;
    for (int device = 0; device < count; ++device) {
        status = hipGetDeviceProperties(&prop, device);
        if (status != hipSuccess) { continue; }
        std::printf("%d.%d ", prop.major, prop.minor);
    }

    return 0;
}
